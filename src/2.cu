#include <mma.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "../include/include.h"
#include <chrono>
#include <hipDNN.h>
#include <vector>

using namespace nvcuda;

#define BLOCK_SIZE_ROW 32
#define BLOCK_SIZE_COL 64
#define HALO 3
#define D_BLOCK_SIZE_COL (BLOCK_SIZE_COL + HALO * 2)
#define D_BLOCK_SIZE_ROW (BLOCK_SIZE_ROW + HALO * 2)
#define PAD 2
#define SM_SIZE_COL (7 * D_BLOCK_SIZE_ROW + PAD)
#define SM_SIZE_ROW (D_BLOCK_SIZE_COL / 8)
#define UNIT_LENGTH 7
#define TENSOR_CORE_M 8
#define IDX(x, y, ldm) ((x) * (ldm) + (y))
#define WARP_PER_BLOCK 8
#define MMA_NUM 13
#define CHANNELS 3


void cuDNN(double** input, const double params[CHANNELS][UNIT_LENGTH * UNIT_LENGTH], double* output, int rows, int cols){
    int H = rows - 2*HALO;
    int W = cols - 2*HALO;

    // Allocate memory on GPU
    double *input_d, *kernel_d, *output_d;
    hipMalloc(&input_d, CHANNELS * rows * cols * sizeof(double));
    hipMalloc(&kernel_d, CHANNELS * UNIT_LENGTH * UNIT_LENGTH * sizeof(double));
    hipMalloc(&output_d, H * W * sizeof(double));

    for (int c = 0; c < CHANNELS; ++c){
        hipMemcpy(input_d + c * rows * cols, input[c], rows * cols * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(kernel_d + c * UNIT_LENGTH * UNIT_LENGTH, params[c], UNIT_LENGTH * UNIT_LENGTH *sizeof(double), hipMemcpyHostToDevice);
        
    }

    // Initialize cuDNN
    hipdnnHandle_t cudnn;
    CHECK_CUDNN(hipdnnCreate(&cudnn));

    hipdnnTensorDescriptor_t input_desc, output_desc;
    hipdnnFilterDescriptor_t kernel_desc;
    hipdnnConvolutionDescriptor_t conv_desc;

    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, CHANNELS, rows , cols));

    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&kernel_desc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(kernel_desc, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, 1, CHANNELS, UNIT_LENGTH, UNIT_LENGTH));

    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, 1, H, W));

    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(conv_desc, 
                                                0, 0, 
                                                1, 1, 
                                                1, 1, 
                                                HIPDNN_CROSS_CORRELATION, 
                                                HIPDNN_DATA_DOUBLE));

    // Select algorithm
    int returned_alog_count;
    hipdnnConvolutionFwdAlgoPerf_t algoPerf;
    CHECK_CUDNN(cudnnGetConvolutionForwardAlgorithm_v7(cudnn,
                                                    input_desc, kernel_desc, conv_desc, output_desc,
                                                    1,
                                                    &returned_alog_count,
                                                    &algoPerf))
    hipdnnConvolutionFwdAlgo_t algo = algoPerf.algo;

    // CHECK_CUDNN(hipdnnFindConvolutionForwardAlgorithm(cudnn,
    //                                                 input_desc, kernel_desc, conv_desc, output_desc,
    //                                                 10, 
    //                                                 &returned_alog_count, 
    //                                                 perf_res));
    // hipdnnConvolutionFwdAlgo_t algo = perf_res[0].algo;

    // hipdnnConvolutionFwdAlgo_t algo;
    // CHECK_CUDNN(hipdnnGetConvolutionForwardAlgorithm)(cudnn,
    //                                                 input_desc, kernel_desc, conv_desc, output_desc,
    //                                                 HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
    //                                                 0,
    //                                                 &algo);

    // Workspace size
    void *workspace = nullptr;
    size_t workspace_size = 0;
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                        input_desc, kernel_desc, conv_desc, output_desc,
                                                        algo, &workspace_size));
    hipMalloc(&workspace, workspace_size);
    
    // Perform convolution
    const double alpha = 1.0f, beta = 0.0f;
    CHECK_CUDNN(hipdnnConvolutionForward(cudnn, &alpha,
                                        input_desc, input_d, kernel_desc, kernel_d, 
                                        conv_desc, algo, 
                                        workspace, workspace_size,
                                        &beta, 
                                        output_desc, output_d));
    
    hipMemcpy(output, output_d, H * W * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(input_d);
    hipFree(kernel_d);
    hipFree(output_d);
    hipFree(workspace);
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyFilterDescriptor(kernel_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroy(cudnn);
}

int main() {
    const int H = 2048, W = 2048;

    int rows = H + 2 * HALO;
    int cols = W + 2 * HALO;
    size_t matrix_size = (unsigned long)rows * cols * sizeof(double);
    
    double **input = (double **)malloc(CHANNELS * sizeof(double *));
    for(int i = 0; i < CHANNELS; i++){
        input[i] = (double *)malloc(matrix_size);
    }

    initialData(input, CHANNELS, rows * cols);

    double params[CHANNELS][UNIT_LENGTH * UNIT_LENGTH];
    for(int c = 0; c < CHANNELS; c++){
        for(int i = 0; i < UNIT_LENGTH * UNIT_LENGTH; i++){
            params[c][i] = 1.0 / (UNIT_LENGTH * UNIT_LENGTH);
        }
    } 

    // -----------------------------------------CUDNN-----------------------------------------------
    std::vector<double> output_cudnn(H * W);
    cuDNN(input, params, output_cudnn.data(), rows, cols);


    printf("output_cudnn\n");
    for(int i = 0; i < 7; i++){
        for(int j = 0; j < 10; j ++){
            printf("%.2f ", output_cudnn[i * W + j]);
        }
        printf("\n");
    }


    for(int c = 0; c < CHANNELS; c++){
        free(input[c]);
    } 
    free(input);
    return 0;
}
